#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, CUDA!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
